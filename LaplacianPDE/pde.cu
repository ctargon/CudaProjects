#include "hip/hip_runtime.h"
#include "pde.h"
#include <math.h>



/*
 

 
 */
__global__ 
void pde(float *U, float *U_out, size_t m, size_t n, size_t length)
{
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;

	if (x < n && y < m)
	{
		float up = down = left = right = 0;

		if (y - 1 >= 0) up = U[(y - 1) * n + x];
		if (y + 1 < m) down = U[(y + 1) * n + x];
		if (x - 1 >= 0) left = U[y * n + (x - 1)];
		if (x + 1 < n) right = U[y * n + (x + 1)];

		U_out[y * n + x] = (up + down + left + right) / 4;
	}
} 

#define TILE_WIDTH 32

__global__ 
void pde_shared(float *U, float *U_out, size_t m, size_t n, size_t length)
{
	// __shared__ unsigned char window[TILE_WIDTH + fWidth - 1][TILE_WIDTH + fWidth - 1];	
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;

	if (x < n && y < m)
	{
		float up = down = left = right = 0;

		if (y - 1 >= 0) up = U[(y - 1) * n + x];
		if (y + 1 < m) down = U[(y + 1) * n + x];
		if (x - 1 >= 0) left = U[y * n + (x - 1)];
		if (x + 1 < n) right = U[y * n + (x + 1)];

		U_out[y * n + x] = (up + down + left + right) / 4;
	}
}


void launch_pde(float *U, float *U_out, size_t m, size_t n, size_t iters)
{
	int k = 0;
	// configure launch params here 
	dim3 block(BLOCK, BLOCK, 1);
	dim3 grid(ceil((float)m / BLOCK), ceil((float)n / BLOCK), 1);

	while (k < iters)
	{
		if (k % 2) pde<<<grid,block>>>(U_out, U, m, n, length);
		else pde<<<grid,block>>>(U_out, U, m, n, length);
		hipDeviceSynchronize();
		checkCudaErrors(hipGetLastError());
	}
}





