#include "hip/hip_runtime.h"
#include "scan.h"
#include <math.h>



/*
 

 
 */
__global__ 
void naive_scan(float *d_in, float *d_out, size_t length)
{
	/*
		Your kernel here: Make sure to check for boundary conditions
	*/
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	int j;

	__shared__ float tmp[BLOCK];

	if (idx < length) tmp[threadIdx.x] = d_in[idx];

	for (j = 1; j <= threadIdx.x; j *= 2)
	{
		__syncthreads();
		float in1 = tmp[threadIdx.x - j];
		__syncthreads();
		tmp[threadIdx.x] += in1;
	}

	if (idx < length) d_out[idx] = tmp[threadIdx.x];
}

__global__
void naive_scan2(float *d_in, float *d_out, size_t length)
{
	int i = threadIdx.x;
	int n = blockDim.x;

	for (int offset = 1; offset < n; offset *= 2)
	{
		float t;
		if (i >= offset) t = d_in[i - offset];
		__syncthreads();
		if (i >= offset) d_in[i] += t;
		__syncthreads(); 
	}
	if (i < length) d_out[i] = d_in[i];
}

__global__
void scan(float *d_in, float *d_out, float *d_sums, size_t length)
{
	__shared__ float temp[2 * BLOCK];
	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	// load shared memory
	if (idx < length / 2) temp[2 * threadIdx.x] = d_in[2 * idx];
	else temp[2 * threadIdx.x] = 0;
	if (2 * idx + 1 < length) temp[2 * threadIdx.x + 1] = d_in[2 * idx + 1];
	else temp[2 * threadIdx.x + 1] = 0;
	__syncthreads();

	for (int stride = 1; stride <= BLOCK; stride *= 2)
	{
		int i = (threadIdx.x + 1) * stride * 2 - 1; // data index
		if (i < 2 * BLOCK)
		{
			temp[i] += temp[i - stride];
		}
		__syncthreads();
	}

	for (int stride = BLOCK / 2; stride > 0; stride /= 2)
	{
		int i = (threadIdx.x + 1) * stride * 2 - 1;
		if (i + stride < 2 * BLOCK)
		{
			temp[i + stride] += temp[i];
		}
		__syncthreads();
	}

	if (idx < length / 2) 
	{
		d_out[2 * idx] = temp[2 * threadIdx.x];
		d_out[2 * idx + 1] = temp[2 * threadIdx.x + 1];
	}

	if (d_sums && threadIdx.x == 0) d_sums[blockIdx.x] = temp[2 * BLOCK - 1];
}

__global__
void increment_scan(float *d_in, float *d_out, float *d_inc, size_t length)
{
	__shared__ float temp[2 * BLOCK];
	__shared__ float increment;

	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	// load shared memory
	temp[2 * threadIdx.x] = d_out[2 * idx];
	temp[2 * threadIdx.x + 1] = d_out[2 * idx + 1];

	if (!threadIdx.x)
	{
		if (blockIdx.x == 0) increment = 0;
		else increment = d_inc[blockIdx.x - 1];
	}
	__syncthreads();

	temp[2 * threadIdx.x] += increment;
	temp[2 * threadIdx.x + 1] += increment; 
	__syncthreads();

	d_out[2 * idx] = temp[2 * threadIdx.x];
	d_out[2 * idx + 1] = temp[2 * threadIdx.x + 1];
}


void launch_scan(float *d_in, float *d_out, float *d_sums, float *d_incs, size_t length)
{
		// configure launch params here 
		dim3 block(BLOCK, 1, 1);
		int grid_d = ceil(length / (2.0 * BLOCK));
		dim3 grid(grid_d, 1, 1);

		scan<<<grid,block>>>(d_in, d_out, d_sums, length);
		hipDeviceSynchronize();
		checkCudaErrors(hipGetLastError());

		int grid_d2 = ceil(grid_d / (2.0 * BLOCK));
		dim3 grid2(grid_d2, 1, 1);
		scan<<<grid2,block>>>(d_sums, d_incs, NULL, length);
		hipDeviceSynchronize();
		checkCudaErrors(hipGetLastError());

		increment_scan<<<grid,block>>>(d_in, d_out, d_incs, length);
		hipDeviceSynchronize();
		checkCudaErrors(hipGetLastError());
}





